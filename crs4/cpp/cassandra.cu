#include <hip/hip_runtime_api.h>
#include "cassandra.h"

namespace other_ns {

template<>
void Cassandra<::dali::GPUBackend>::RunImpl(::dali::DeviceWorkspace &ws) {
  const auto &input = ws.Input<::dali::GPUBackend>(0);
  const auto &shape = input.shape();
  auto &output = ws.Output<::dali::GPUBackend>(0);
  for (int sample_idx = 0; sample_idx < shape.num_samples(); sample_idx++) {
    CUDA_CALL(hipMemcpyAsync(
            output.raw_mutable_tensor(sample_idx),
            input.raw_tensor(sample_idx),
            shape[sample_idx].num_elements() * input.type_info().size(),
            hipMemcpyDeviceToDevice,
            ws.stream()));
  }
}

}  // namespace other_ns

DALI_REGISTER_OPERATOR(Crs4Cassandra, ::other_ns::Cassandra<::dali::GPUBackend>, ::dali::GPU);
